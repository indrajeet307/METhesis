#include "helper.h"
#include "naiveOperations.h"
#include "trie.h"
#include <assert.h>
#include <errno.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>

int readFromTestFile(char *fname, s_docs **docptr, s_trie **classptr, s_trie **wordptr)
{
    FILE *fp;
    int err;
    errno = 0;
    fp = fopen(fname, "r");
    err = errno;
    if(fp == NULL)
    {
        printf("[ Error ] %s.\n",strerror(err));
        return err;
    } 
    char *buff=NULL;
    int numWords=0;
    size_t count=0;
    size_t readlen=0;
    while( !feof(fp))
    {
        readlen = getline( &buff, &count, fp);
        if( readlen == -1) break;
        char *class = strtok(buff,"\t");
        int cid = findWord(classptr, class, strlen(class));
        if( cid < 0) 
        {
            printf(" Found an Outlier Class %s\n", class);
            continue;
        }
        s_doc *temp = addDocTolist(docptr,  cid);

        char *word;
        while( (word = strtok(NULL," \n\r")) != NULL)
        {
            numWords++;
            int wid = findWord(wordptr, word, strlen(word));
            if( wid < 0)
            {
                //printf(" New Word %s, cannot accomodate\n", word);
                continue;
            }
            addWordDoc(&temp,docptr,wid);
        }
    }
    if ( buff !=NULL)
        free(buff);
    printf("Number of words %d\n",numWords);
    fclose(fp);
    return 0;
}

int readFromTrainFile(char *fname, s_docs **docptr, s_trie **classptr, s_trie **wordptr)
{
    FILE *fp;
    int err;
    errno = 0;
    fp = fopen(fname, "r");
    err = errno;
    if(fp == NULL)
    {
        printf("[ Error ] %s.\n",strerror(err));
        return err;
    } 
    char *buff=NULL;
    size_t count=0;
    size_t readlen;
    int numWords=0;
    while( !feof(fp))
    {
        readlen = getline( &buff, &count, fp);
        if( readlen == -1) break;
        char *class = strtok(buff,"\t");
        int cid = findWord(classptr, class, strlen(class));
        if( cid < 0)
            cid = addWord(classptr, class, strlen(class));
        s_doc *temp = addDocTolist(docptr,  cid);

        char *word;
        while( (word = strtok(NULL," \n\r")) != NULL)
        {
            numWords++;
            int wid = findWord(wordptr, word, strlen(word));
            if( wid < 0)
                wid = addWord(wordptr, word, strlen(word));
            addWordDoc(&temp,docptr,wid);
        }
    }
    if ( buff !=NULL)
        free(buff);
    printf("Number of words %d\n",numWords);
    fclose(fp);
    return 0;
}

int* createMatrix( int columns, int rows)
{
    int *temp = (int*) calloc( sizeof(int), columns*rows);
    return temp;
}

int *createVector(int columns)
{
    int *temp = (int*) calloc( sizeof(int), columns);
    return temp;
    }

float* createFloatMatrix( int columns, int rows)
{
    float *temp = (float*) calloc( sizeof(float), columns*rows);
    return temp;
}

float getAccuracy(int *pmat, int *cvect, int total)
    {
        int i;
        float ans=0.0;
        for ( i=0; i<total; i++)
        {
            if( pmat[i] == cvect[i])
                ans ++;
        }
        return ans/total;
    }

int main(int argc, char **argv)
{
    if( argc < 2)
    {
        printf("Usage: ./naive.out <training_file> <test_file>\n");
        return 1;
    }
    char * trainfname = argv[1];
    char * testfname = argv[2];

    s_docs *doclist = initDL();
    s_docs *testdoclist = initDL();
    s_trie *classlist = initTrie();
    s_trie *wordlist = initTrie();


    readFromTrainFile(trainfname, &doclist, &classlist, &wordlist);
    int numdocs = getNumDocs(doclist);
    int numclasses = getNumwords(classlist);
    printf(" Read %d Docs\n", numdocs);
    printf(" Containing %d total unique words\n", getNumwords(wordlist));
    int numwords = getNumwords(wordlist);

    int i=0;
    int count=0;
    int * mat = createMatrix( numwords, numdocs); // extra column for the classID
    assert( mat != NULL );
    int * cvect = createVector( numdocs);    // stores the class info for the doc
    assert( cvect != NULL );
        s_doc *docptr = doclist->list;

    count = filldata(doclist, mat, cvect ,numdocs, numwords);
    //printIntMatrix(mat, numdocs, numwords);
    float *fmat = createFloatMatrix( numdocs, numwords);
    normalize(mat ,fmat, numdocs, numwords);
    //print(fmat, numdocs, numwords);
    float * cprob = (float*) calloc( sizeof(float) , numclasses);
    assert( cprob != NULL );
    float * probmat=createFloatMatrix( numwords, numclasses); // extra column for probablity of the class
    createProbablityMatrix( mat, probmat, cvect, cprob, numdocs, numwords, numclasses, numwords);
    //print(probmat, numclasses, numwords);

    free(cvect);
    readFromTestFile( testfname, &testdoclist, &classlist, &wordlist);
    numdocs = getNumDocs(testdoclist);
    printf(" Read %d Training Docs\n", numdocs);
    printf(" Containing %d total unique words\n", getNumwords(wordlist));
    int *smat = createMatrix( numwords, numdocs); 

     cvect = createVector( numdocs);    // stores the class info for the doc
    assert( cvect != NULL );

    filldata(testdoclist, smat, cvect, numdocs, numwords);
    int * pmat = createVector( numdocs); // for predicted classID
    
    assignClass( smat, probmat, cprob, pmat, numdocs, numclasses, numwords); 
    //printIntMatrix( pmat, 1, numdocs);
    //printIntMatrix( cvect, 1, numdocs);
    float acc = getAccuracy(pmat, cvect, numdocs)*100;
    printf(" Accuracy is %f %%.\n",acc);
    showWords(classlist);

    free(cprob);
    free(cvect);
    free(fmat);
    free(mat);
    free(pmat);
    free(probmat);
    free(smat);
    cleanUP(&doclist);
    cleanUP(&testdoclist);

    deleteTrie( &classlist);
    deleteTrie( &wordlist);
    free( doclist);
    free( testdoclist);
    return 0;
}
