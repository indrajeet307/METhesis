
#include <hip/hip_runtime.h>
#include <assert.h>
#include <limits.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

/*!
 *	\brief calculate normalized frequency matrix 
 *
 *   Normalized value(x) = \f$ \frac{ X - min }{ max - min }\f$
 *	\return normalized matrix
 *	
 */
int normalize( 
        int *in_mat,     /*!< [in] input matrix */
        float *out_mat,     /*!< [out] output matrix */
        int in_rows,     /*!< [in] number of rows in the input matrix */
        int in_columns    /*!< [in] number of columns in the output matrix */
        )
{
    int i,j;
    int max, min;
    for ( i=0; i<in_rows; i++)
    {
        max = INT_MIN;
        min = INT_MAX;
        for ( j=0;j<in_columns; j++)
        {
            if( min > in_mat[ (i*in_columns)+j ] )
                min = in_mat[ (i*in_columns)+j ] ;

            if( max < in_mat[ (i*in_columns)+j ] )
                max = in_mat[ (i*in_columns)+j ];
        }
        for (  j=0; j<in_columns; j++)
        {
            out_mat[(i*in_columns)+j] =(float)(in_mat[(i*in_columns)+j] - min)/(float)(max-min)   ;
        }
    }
    return 0;
}

/*!
 *	\brief Calculates the total frequency of each file for each class
 *
 *	function goes through all the files, calculates total frequency of each total 
 *	frequency for each file is stored in last column of the output matrix
 *	in_mat:
 *	    numfiles X totalnumopcodes matrix, containing frequency of each opcode
 *	out_mat:
 *	    numclass X totalnumopcodes+1 matrix, containing total frequency of each opcode and
 *	    each file in last column
 *	in_inrows:
 *	    = numfiles
 *	in_incolumns:
 *	    = totalnumopcodes
 *	in_outrows:
 *	    = number of classes 
 *
 *	\return modified outmat contains total frequency for each opcode and each file
 *	
 */
int createClassWiseData( 
        int *in_mat,     /*!< [in] input matrix */
        int *out_mat,     /*!< [out] output matrix */
        int in_rows,     /*!< [in] number of rows in input matrix */
        int in_columns,     /*!< [in] number of columns in input matrix */
        int in_outrows    /*!< [in] number of rows in output matrix */
        )
{
    int i,j;
    int sum=0;
    // for each file
    for (  i=0; i<in_rows; i++)
    {
        sum = 0;
        // for each opcode
        for ( j=0; j<in_columns-1; j++)
        {
            // add the frequency for each opcode
            out_mat[ (i*in_columns)+j ] += in_mat [ (i*in_columns)+j ];
            // save the frequency for total collection
            sum += in_mat[ (i*in_columns)+j ];
        }
        out_mat[j] = sum;
    }
    return 0;
}

/*!
 *	\brief gives probability of occurrence of each opcode in each class
 *
 *   takes as input a matrix containing normalized frequencies of each opcode in a file, 
 *   probability of opcode occurring in a class is found, by adding frequencies of each
 *   opcode in each file of each class and then dividing by sum of all frequencies in that
 *   class. We take log so the values are in a smaller range
 *
 *	in_inmat:
 *	    numfiles X totalnumopcodes matrix, has normalized frequency value for each opcode
 *
 *	in_cvect:
 *	    numfiles in length, contains,class ,the current row in in_inmat belongs to
 *
 *	in_inrows:
 *	    = numfiles
 *
 *	in_incolumns:
 *	    = totalnumopcodes
 *
 *	in_outrows:
 *	    = number of classes 
 *
 *	in_outcolumns:
 *	    = totalnumopcodes
 *
 *	out_outmat:
 *	    in_outrows X in_outcolumns, probability of each opcode in each class
 *
 *	out_cprob:
 *	    probability of each class
 *
 *	\return out_outmat is the modified matrix which contains the required probabilities
 *	
 */
int createProbablityMatrix( 
        int     *in_inmat,    /*!< [in] input matrix */
        float   *out_outmat,    /*!< [out] output matrix */
        int     *in_cvect,    /*!< [in] class vector */
        float   *out_cprob,    /*!< [out] probability of each class */
        int     in_inrows,    /*!< [in] number of rows in input matrix */
        int     in_incolumns,    /*!< [in] number of columns in input matrix */
        int     in_outrows,    /*!< [in] number of rows in output matrix */
        int     in_outcolumns    /*!< [in] number of columns in output matrix */
        )
{
#define  INDEX(i,j,cols) ((i*cols)+j)
    int i,j;
    int cls;
    float *class_wise_total=(float*) calloc(sizeof(float),in_outrows);
    assert(class_wise_total != NULL );
    // For each file
    for ( i=0; i<in_inrows; i++)
    {
        // get class for current file
        cls = in_cvect[ i ];
        // For each opcode
        for ( j=0; j<in_outcolumns; j++)
        {
            // add frequency of each opcode to appropriate a class opcode
            out_outmat[ INDEX(cls,j,in_outcolumns) ] +=   (float)in_inmat[ INDEX(i,j,in_incolumns) ];
            // add frequency to current class frequency
            class_wise_total [ cls ] +=   (float)in_inmat[ INDEX(i,j,in_incolumns) ];
        }
        // increase the count of file in current class
        out_cprob[ cls ] += 1;
    }
    // For each class
    for ( i=0; i<in_outrows; i++)
    {
        // For each opcode
        for ( j=0; j<in_outcolumns; j++)
        {
            // store the probability of current opcode
            float temp = (log10((out_outmat[ INDEX(i,j,in_outcolumns) ]+1) / (class_wise_total[ i ]+1) ));
            out_outmat[ INDEX(i,j,in_outcolumns) ] = (-1)*temp; // multiply by -1, because log[0-1] < 0
        }
        // save probability of current class
        out_cprob[ i ] = (-1)*log10(out_cprob[ i ]/ in_inrows); // multiply by -1 because log[0-1] < 0
    }
    free(class_wise_total);
    return 0;
#undef INDEX
}

/*!
 *	\brief Displays the float matrix
 *
 *   Should not be used on huge matrices
 *
 *	\return 
 *	
 */
void printFloatMatrix(
        float *in_mat,     /*!< [in] pointer to the matrix */
        int in_rows,     /*!< [in] number of rows in the matrix */
        int in_columns    /*!< [in] number of columns in the matrix */
        )
{
    int i,j;
    for ( i=0; i<in_rows; i++)
    {
        for ( j=0; j<in_columns; j++)
        {
            printf(" %f",in_mat[ (i*in_columns)+j ]);
        }
        printf("\n");
    }
}

/*!
 *	\brief Displays the int matrix
 *
 *   Should not be used on huge matrices
 *
 *	\return 
 *	
 */
void printIntMatrix( 
        int *in_mat,     /*!< [in] pointer to the matrix */
        int in_rows,     /*!< [in] number of rows in the matrix */
        int in_columns    /*!< [in] number of columns in the matrix */
        )
{
    int i,j;
    for ( i=0; i<in_rows; i++)
    {
        for ( j=0; j<in_columns; j++)
        {
            printf(" %d",in_mat[ (i*in_columns)+j ]);
        }
        printf("\n");
    }
}

/*!
 *	\brief Assigns class to all the test files
 *
 *   takes a matrix of normalized frequency values for each test file, and probability
 *   matrix for each opcode of each class, calculates probability of each opcode for each
 *   class, assigns a class to a file with maximum probability
 *
 *   in_mat:
 *       numfiles X total_number_of_opcodes matrix, containing normalized frequencies
 *
 *   in_cprob:
 *       numclass wide vector, containing probability of each class
 *
 *   out_pridict:
 *       numfiles wide vector, containing assigned class for each file
 *
 *   in_rows:
 *       numfiles
 *
 *   in_classes:
 *       numclasses
 *
 *   in_columns:
 *       total number of opcodes
 *
 *	\return predicted class vector
 *	
 */
void assignClass( 
        int *in_mat,     /*!< [in] input matrix */
        float *in_prob,     /*!< [in] probability matrix */
        float *in_cprob,     /*!< [in] class probability matrix */
        int *out_pridict,     /*!< [out] predicted class */
        int in_rows,     /*!< [in] number of input rows */
        int in_classes,     /*!< [in] number of classes */
        int in_columns    /*!< [in] number of columns */
        )
{
#define  INDEX(i,j,cols) ((i*cols)+j)
    int i,j,k;
    double *classprob =  (double*) calloc( sizeof(double), in_classes);
    // for each file
    for ( i=0; i<in_rows; i++)
    {
        // for each class
        for ( k=0; k<in_classes; k++)
            classprob[ k ] = in_cprob[k];
        // for each opcode
        for ( j=0; j<in_columns; j++)
        {

            // for each class
            for ( k=0; k<in_classes; k++)
            {
                // for opcodes having normalized frequency greater than 1
                if ( in_mat [ INDEX(i,j,in_columns) ] > 0 ) 
                {
                    // add the probability to current class ( add because we already have
                    // log10 of those values)
                    // TODO remove multiplication and check
                    classprob[ k ] += in_mat [ INDEX(i,j,in_columns) ]*in_prob [ INDEX(k,j,in_columns) ];
                }
            }
        }
        int maxClass=0;
        // for each class 
        for ( k=0; k<in_classes; k++)
        {
            // save the max class
            if( classprob[ maxClass ] > classprob[k] )
                maxClass = k;
        }
        // assign the max class
        out_pridict[i] = maxClass;
    }
    free(classprob);
#undef INDEX
}

/*!
 *	\brief creates a int matrix of row X column dimensions
 *
 *   This is actually a vector of size rows X columns X sizeof(int)
 *   use the traditional way for accessing the vector elements
 *
 *	\return pointer to the allocated matrix
 *	
 */
int* createIntMatrix( 
        int in_rows,      /*!< [in] number of rows in the matrix */
        int in_columns    /*!< [in] number of columns in the matrix */
        )
{
    int *temp = (int*) calloc ( sizeof(int) , in_rows*in_columns );
    return temp;
}

/*!
 *	\brief creates a float matrix of row X column dimensions
 *
 *   This is actually a vector of size rows X columns X sizeof(float)
 *   use the traditional way for accessing the vector elements
 *
 *	\return pointer to the allocated matrix
 *	
 */
float* createFloatMatrix(
        int in_rows,      /*!< [in] number of rows in the matrix */
        int in_columns    /*!< [in] number of columns in the matrix */
        )
{
    float *temp = (float*) calloc( sizeof(float), in_columns*in_rows);
    return temp;
}

/*!
 *	\brief creates a integer vector of length size
 *
 *   More Details ...
 *	\return pointer to the newly created vector
 *	
 */
int *createVector(
        int in_size      /*!< [in] length of the vector */
        )    
{
    int *temp = (int*) calloc( sizeof(int), in_size);
    return temp;
}


/*!
 *	\brief Gives accuracy of current configuration
 *
 *   More Details ...
 *
 *	\return ratio of correct predictions to number of predictions
 *	
 */
float getAccuracy( 
        int *in_pmat,   /*!< [in] predicated class vector */
        int *in_cvect,  /*!< [in] actual class vector */
        int in_total    /*!< [in] length of the vector */
        )
{
    int i;
    float ans=0.0;
    for ( i=0; i<in_total; i++)
    {
        if( in_pmat[i] == in_cvect[i])
            ans ++;
    }
    return ans/in_total;
}

/*!
 *	\brief  Gives transpose of a matrix
 *
 *   The rows of in_mat are converted to columns of the out_mat, and columns of in_mat are
 *   converted to rows of out_mat
 *                                                                                       
 *	in_mat    =   in_rows      X   in_columns
 *
 *	out_mat   =   in_columns   X   in_rows
 *
 *	\return transpose of a matrix in out_mat
 *	
 */
void rotateMatrix( 
        int *in_mat,     /*!< [in] input matrix */
        int *out_mat,    /*!< [out] output matrix */
        int in_rows,     /*!< [in] number of rows in input matrix */
        int in_columns   /*!< [in] number of columns in output matrix */
        )
{
    int outcolumns=in_rows;
    int i,j;

    for ( i=0; i<in_rows; i++)
    {
        for ( j=0;j<in_columns;j++)
        {
            out_mat[ j*outcolumns + i ] = in_mat[ i*in_columns+j ];
        }
    }
}
void rotateMatrixF( 
        float *in_mat,     /*!< [in] input matrix */
        float *out_mat,    /*!< [out] output matrix */
        int in_rows,     /*!< [in] number of rows in input matrix */
        int in_columns   /*!< [in] number of columns in output matrix */
        )
{
    int outcolumns=in_rows;
    int i,j;

    for ( i=0; i<in_rows; i++)
    {
        for ( j=0;j<in_columns;j++)
        {
            out_mat[ j*outcolumns + i ] = in_mat[ i*in_columns+j ];
        }
    }
}

/*!
 *	\brief Gives the probablity for the current in_val
 *
 *   probability = \f$ \frac{1}{\sqrt{2\Pi\sigma^{2}}}\exp^{\frac{(x-\mu)^{2}}{2\sigma^{2}}} \f$
 *
 *	\return probablity in float
 *	\see 
 *	
 */
float getTheProbablity( 
        float in_vval,  /*!< [in] x as in above formulae */
        float in_vmean, /*!< [in] mean value */
        float in_vvar   /*!< [in] variance value */
        )
{

    float result=0.0;
    float val1 =  1/sqrt( 2.0* M_PI* in_vvar);
    float val2 = (in_vval-in_vmean)*(in_vval-in_vmean)/(2.0*in_vvar);
    val2 = 1 / exp( val2);
    result = log10( val1*val2);
    if( isnan(result) || isinf(result) ) return 0.0;
    return result;
}

/*!
 *	\brief Assigns class to the test inputs
 *
 *   For all the test files in testArray, gets the group index from in_group_index( which
 *   is decided based on file size), selects the probablites from in_trainMatrix only for
 *   those opcodes whose normalized occurances are greater than 0
 *   
 *   in_trainMatrix:
 *     four rows in in_trainMatrix are considered as one row, structure is as follows
 *
 *   |               |         |          | in_num_opcodes number of columns |
 *   |:-------------:|:-------:|:--------:|:--------------------------------:|
 *   | group [0-99]  | benign  | mean     |                                  |
 *   | in_num_groups |         | variance |                                  |
 *   |               | malware | mean     |                                  |
 *   |               |         | variance |                                  |
 *
 *    in_testMatrix:
 *   |                 | in_num_opcodes number of columns |
 *   |:---------------:|:--------------------------------:|
 *   | inumber of rows=n_numtestfiles  ||
 *
 *   in_group_index:
 *   has in_numtestfiles number of values containg group number of file, can be thought of
 *   extra column in in_testMatrix 
 *
 *   \return predicted class for files in in_testMatrix
 *	  
 */
void assignClassUsingMeanVarianceData(
        float *in_trainMatrix, /*!< [in] trained probablity matrix */
        float *in_testMatrix,  /*!< [in] testing matrix */
        int in_num_groups,     /*!< [in] number of groups / number of rows in train matrix */
        int in_num_opcodes,    /*!< [in] number of opcodes / number of columns in test,train matrix */
        int in_numtestfiles,   /*!< [in] number of test files / number of rows in test matrix */
        int *in_group_index,   /*!< [in] vector containing group index of each file in test matrix( 1:1 mapping) */
        int *out_predict_vect  /*!< [out] predicted class  */
        )

{
    int i,j;
    float pmal=0.0, pben=0.0;
    int mean =0, var =1; /// \todo make this genric or avoid it somehow
    float vmean =0, vvar =1;
    int index=0;
    // Iterate through each file in in_testMatrix
    for ( i=0; i<in_numtestfiles; i++)
    {
        index = in_group_index[i]*4;
        pmal=0.0; 
        pben=0.0;
        // For all the opcodes
        for ( j=0; j<in_num_opcodes; j++)
        {
            // If any opcode in current file has normalized freq > 0
            if( in_testMatrix[i*in_num_opcodes+j] > 0 ) 
            {
                // get vmean and var considering it is a benign
                vmean = in_trainMatrix[(index+0+mean)*in_num_opcodes+j];
                vvar  = in_trainMatrix[(index+0+var )*in_num_opcodes+j];
                assert( vmean >= 0.0f && vvar >= 0.0f );
                // add the probablity in benign
                pben += getTheProbablity( in_testMatrix[i*in_num_opcodes+j], vmean, vvar);

                // get vmean and var considering it is a malware
                vmean = in_trainMatrix[(index+2+mean)*in_num_opcodes+j];
                vvar  = in_trainMatrix[(index+2+var )*in_num_opcodes+j];
                assert( vmean >= 0.0f && vvar >= 0.0f );
                // add the probablity in malware
                pmal += getTheProbablity( in_testMatrix[i*in_num_opcodes+j], vmean, vvar);
            }
        }

        // assign class depending on which probability is higher
        if( pmal > 0 && pben > 0)
            out_predict_vect[i] = pmal > pben ? 1 : 0;
        else
        {
            if( pmal > 0 ) out_predict_vect[i] = 1;
            else if ( pben > 0 ) out_predict_vect[i] = 0;
            else //( pmal < 0 && pben < 0)
                out_predict_vect[i] = pmal > pben ? 1 : 0;

        }
    }
}

/*!
 *	\brief This function assigns class based on the selective features
 *
 *   Most of the details similar to function assignClassUsingMeanVarianceData, except for
 *   addition of in_feature list
 *
 *   in_feature_list:
 *   contains list of prominant features for each group
 * |                         | each row points to the feature vector in s_group   |
 * | :----------------------:| :------------------------------------------------: |
 * | rows = number of groups |                                                    |
 *
 *   the feature vector is in_num_opcodes in len, vector has bit set only if the opcode
 *   having id = index of this vector is amongst the prominant feature
 *
 *   \todo huge dependency !!! try and remove it viz feature list is pointing to feature vector
 *   in s_group
 *
 *	\return predicted class matrix
 *	
 */
void assignClassUsingMeanVarianceDataAndFeatureSelection(
        float *in_trainMatrix,  /*!< [in] trained probability matrix */
        float *in_testMatrix,   /*!< [in] testing matrix */
        int **in_feature_list,  /*!< [in] array of list of feature vector for each group, number of lists = number of groups */
        int in_num_groups,      /*!< [in] number of groups / number of rows in train matrix */
        int in_num_opcodes,     /*!< [in] number of opcodes / number of columns in test,train matrix */
        int in_numtestfiles,    /*!< [in] number of test files / number of rows in test matrix */
        int *in_group_index,    /*!< [in] vector containing group index of each file in test matrix( 1:1 mapping) */
        int *out_predict_vect   /*!< [out] predicted class  */
        )
{
    int i,j;
    float pmal=0.0, pben=0.0;
    int mean =0, var =1; /// \todo make this genric or avoid it somehow
    float vmean =0, vvar =1;
    int index=0;
    int grpindex=0;
    // for each file in in_testMatrix
    for ( i=0; i<in_numtestfiles; i++)
    {
        grpindex = in_group_index[i];
        index = grpindex*4;
        pmal=0.0; 
        pben=0.0;
        // for each opcode
        for ( j=0; j<in_num_opcodes; j++)
        {
            // if the normalized frequency in in_testMatrix is greater than 0 and current
            // opcode is amongst the prominent opcode for the group,to which the current
            // file belongs.
            if( in_testMatrix[i*in_num_opcodes+j] > 0 && in_feature_list[grpindex][j] ) 
            {
                // get variance and mean assuming it is a benign file
                vmean = in_trainMatrix[(index+0+mean)*in_num_opcodes+j];
                vvar  = in_trainMatrix[(index+0+var )*in_num_opcodes+j];
                assert( vmean >= 0.0f && vvar >= 0.0f );
                // add to probability of file being benign
                pben += getTheProbablity( in_testMatrix[i*in_num_opcodes+j], vmean, vvar);

                // get variance and mean assuming it is a malware file
                vmean = in_trainMatrix[(index+2+mean)*in_num_opcodes+j];
                vvar  = in_trainMatrix[(index+2+var )*in_num_opcodes+j];
                assert( vmean >= 0.0f && vvar >= 0.0f );
                // add to probability of file being malware
                pmal += getTheProbablity( in_testMatrix[i*in_num_opcodes+j], vmean, vvar);
            }
        }
        /// assign class whos probablity is greater
        if( pmal > 0 && pben > 0)
            out_predict_vect[i] = pmal > pben ? 1 : 0;
        else
        {
            if( pmal > 0 ) out_predict_vect[i] = 1;
            else if ( pben > 0 ) out_predict_vect[i] = 0;
            else //( pmal < 0 && pben < 0)
                out_predict_vect[i] = pmal > pben ? 1 : 0;

        }
    }
}
