#include "hip/hip_runtime.h"
#include "opcodeFile.h"
#include "trie.h"
#include <assert.h>
#include <errno.h>
#include <limits.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

// TODO update all enums with
// http://stackoverflow.com/questions/1102542/how-to-define-an-enumerated-type-enum-in-c
// TODO Garbage Collection
enum classId{ BENIGN=0, MALWARE=1, UNKNOWN };
int getClassId( 
        char * cls
        )
{
    if ( strcmp( cls, "MALWARE") == 0 ) return MALWARE;
    if ( strcmp( cls, "BENIGN") == 0 ) return BENIGN;
    return -1;
}

void initFiles( 
        s_files ** p_files
        )
{
    *p_files = (s_files*) malloc ( sizeof(s_files) );
    (*p_files)->list = NULL;
    (*p_files)->garb = NULL;
}

void initFileList( 
        s_filelist ** p_list
        )
{
    (*p_list) = (s_filelist*) malloc ( sizeof( s_filelist) ) ;
    (*p_list)->count = 0;
    (*p_list)->list = NULL;
}

void initGroups( 
        s_group ** out_groups, 
        int in_count
        )
{
    (*out_groups) = (s_group*) calloc(sizeof(s_group), in_count);
    int i;
    for ( i=0; i<in_count; i++)
    {
        (*out_groups)[i].list[0] = NULL;
        (*out_groups)[i].list[1] = NULL;
        (*out_groups)[i].features = NULL;
    }     
}

s_fileProp* createFileNode( 
        char *filename, 
        int filesize, 
        char *data_type, 
        char *cls, 
        int numopcode, 
        int totalopcodes
        )
{
    int err =0;
    s_fileProp* temp = (s_fileProp*) malloc( sizeof(s_fileProp) );
    err = errno;
    if( temp == NULL || errno )
    {
        printf(" Malloc Failed %s\n", strerror(err));
        exit(1);
        return NULL;
    }
    temp->name = (char*) malloc( sizeof(char) * (strlen(filename)+1) );
    strcpy( temp->name ,filename);
    temp->numopcode = numopcode;
    temp->size = filesize;
    temp->classId   = getClassId(cls);
    temp->opcodes = (s_opcodenode*) calloc ( sizeof(s_opcodenode) , numopcode);
    temp->normalized_opcodes = (float*) calloc ( sizeof(float) , numopcode);
    return temp;
}

void createFeatureListForEachGroup( 
        int ***out_feature_list, 
        int in_num_groups
        )
{
    (*out_feature_list) = (int**) calloc( sizeof(int*), in_num_groups);
}

void addToFiles(
        s_files ** p_files, 
        s_fileProp ** p_fileprop
        )
{
    (*p_files)->numFiles ++;
    if( (*p_files)->list== NULL )
    {
        (*p_files)->list = (*p_fileprop);
        return;
    }
    (*p_fileprop)->next = (*p_files)->list;
    (*p_files)->list = (*p_fileprop);
    return;
}

void addToList( 
        s_filelist ** p_list, 
        s_fileProp * p_prop
        )
{
    s_filelistnode *temp = NULL;
    temp = (s_filelistnode*) malloc (sizeof(s_filelistnode) );
    temp->prop = p_prop;
    temp->next = NULL;
    if( (*p_list)->list == NULL)
    {
        (*p_list)->list =temp;
        (*p_list)->count+=1;
    }
    temp->next = (*p_list)->list;
    (*p_list)->list = temp;
    (*p_list)->count+=1;
}

void addToGroup( 
        s_group ** out_groups, 
        int in_gropup_index, 
        s_fileProp *in_fileprop
        )
{
    s_filelistnode *temp,*new_node;
    s_group *currgrp;
    int currClassId = in_fileprop->classId;

    currgrp =&( (*out_groups)[in_gropup_index] );

    temp = (*out_groups)[in_gropup_index].list[  currClassId ];

    new_node = (s_filelistnode*) malloc (sizeof(s_filelistnode) );
    new_node->prop = in_fileprop;
    new_node->next = NULL;

    if( temp == NULL )
    {
        currgrp->list[ currClassId ] = new_node;
        currgrp->list[ currClassId ]->next  = NULL;

        currgrp->count = 1;
        currgrp->max = in_fileprop->numopcode;
        currgrp->min = in_fileprop->numopcode;

        return;
    }
    new_node->next = currgrp->list[ currClassId ];
    currgrp->list[ currClassId ] = new_node;

    currgrp->count += 1;
    currgrp->max = (in_fileprop->numopcode > currgrp->max) ? in_fileprop->numopcode : currgrp->max ;
    currgrp->min = (in_fileprop->numopcode < currgrp->min) ? in_fileprop->numopcode : currgrp->min ;
}

int readOpcodeFile( 
        char* fname, 
        s_trie** opcodelist
        )
{
    FILE *fp;
    int err;
    int numopcodes=0;
    err = errno = 0;
    fp = fopen( fname, "r");
    err = errno;
    if( fp == NULL )
    {
        printf("[ Error ] %s.\n", strerror(err));
        return -1;
    }
    char  *buff=NULL;
    //char  *opcode=NULL;
    size_t count=0;
    size_t readlen=0;
    while( !feof( fp ) )
    {
        readlen = getline( &buff, &count, fp);
        if( readlen == (size_t)-1 ) break ;
        //opcode = strtok( buff, "\n" ); // TODO construct a opcode word list
        numopcodes++;
    }
    fclose( fp );
    return numopcodes;
}

s_fileProp * convertLineToFileProp(
    char *buff,
    int *out_groupcount,
    int in_totalnumopcodes,
    int *out_numFiles
)
{
        char *filename = strtok( buff, "," );
        char *filesize = strtok( NULL, "," );
        char *data_set = strtok( NULL, "," );
        char *cls    = strtok( NULL, "," );
        char *numopc   = strtok( NULL, "," );
        int numopcode = atoi( numopc );
        int size = atoi( filesize );
        int index = size/5;
        int max = INT_MIN;
        int min = INT_MAX;

        if( numopcode > 10 && size < 500 )
        {
            out_groupcount[ index*2 + getClassId(cls) ] += 1; // TODO replace 2 with NUM_CLASSES
            s_fileProp *tempfile = createFileNode( filename, size, data_set, cls,
            numopcode, in_totalnumopcodes);
            char *freq;
            int numopcodes=0;
            int id=0;
            while( (freq = strtok(NULL,",\n")) != NULL )
            {
                int currfreq = atoi(freq);
                if( currfreq > 0)
                {
                    if (  currfreq> max ) max = currfreq;
                    if ( currfreq< min ) min = currfreq;
                    tempfile->opcodes[ numopcodes ].id = id;
                    tempfile->opcodes[ numopcodes++ ].freq = currfreq ;
                    assert( atoi(freq) > 0 );
                }
                id++;
            }
            // THIS IS REALLY WEIRED NEED TO DO SOMETHING ABOUT THIS
            if( numopcodes < numopcode  ) // Just in case the CSV  as an opcode 
            {
                tempfile->numopcode = numopcodes;
            }
            tempfile->min_opcodefreq = min;
            tempfile->max_opcodefreq = max;
            tempfile->next = NULL;
            (*out_numFiles)++;
            return tempfile;
        }
        return NULL;
}
int readCSVFile( 
        char* in_filename, 
        int in_numopcode, 
        s_files ** out_fillist, 
        int *out_groupcount
        )
{
    FILE *fp;
    int err;
    int numfiles=0;
    err = errno = 0;
    fp = fopen( in_filename, "r");
    err = errno;
    if( fp == NULL )
    {
        printf("[ Error ] %s.\n", strerror(err));
        return -1;
    }
    size_t count=0;
    char  *buff=NULL;

    while( getline(&buff, &count, fp) > 0 )
    {
        s_fileProp *tempfile = convertLineToFileProp( buff, out_groupcount, in_numopcode, &numfiles);
        if( tempfile != NULL)
            addToFiles( out_fillist, &tempfile);
    }
    free( buff );
    fclose( fp );
    return numfiles;
}

void cleanUp( 
        s_garbage * p_garbage
        )
{
}

void deleteFiles( 
        s_files ** p_files
        )
{
    cleanUp( (*p_files)->garb );
    free( *p_files );
    (*p_files) = NULL;
}


void fillTheMatrix( 
        s_files ** p_files, 
        int * p_mat, 
        int * p_cvect, 
        int rows, 
        int columns
        )
{
    s_fileProp * list = (*p_files)->list;
    int i=0,j=0;
    while( list!= NULL)
    {
        for( j=0; j<list->numopcode; j++)
        {
            p_mat[ (i*columns)+ list->opcodes[j].id ] = list->opcodes[j].freq;
        }
        p_cvect[i] = list->classId;
        list= list->next;
        i++;
    }
}

void fillTheMatrixFromList( 
        s_filelist ** p_files, 
        int * p_mat, 
        int * p_cvect, 
        int rows, 
        int columns
        )
{
    s_filelistnode * list = (*p_files)->list;
    int i=0,j=0;
    while( list!= NULL && i<rows)
    {
        for( j=0; j<list->prop->numopcode; j++)
        {
            p_mat[ (i*columns)+ list->prop->opcodes[j].id ] = list->prop->opcodes[j].freq;
        }
        p_cvect[i] = list->prop->classId;
        list= list->next;
        i++;
    }
}


int adjustCountInEachGroup(
        int* out_groupcount, 
        int num_groups
        )
{
    int i;
    int count=0;
    for(  i=0; i<num_groups*2; i+=2)
    {
        if( out_groupcount[ i+0 ] > out_groupcount[ i+1 ])
        {
            out_groupcount[ i+0 ] = out_groupcount[ i+1 ];
            count += out_groupcount[i+0];
        }
        else
        {
            out_groupcount[ i+1 ] = out_groupcount[ i+0 ];
            count += out_groupcount[i+1];
        }
    }
    return count*2;
}

/*
 *	@DESC   : Does grouping for only 2 classes
 *          : TODO make it genric for n classes
 *	@PRAM   : What are the parameters?
 *	@RETURN : What does it return?
 *	
 */
void doGrouping( 
        s_files* in_files, 
        int* in_groupcount, 
        s_group ** out_groups
        )
{
    s_fileProp *temp = in_files->list;
    int groupIndex=0;
    int classId=-1;
    int index=0;
    while( temp != NULL )
    {
        groupIndex = temp->size/5;
        classId = temp->classId;
        index = groupIndex*2+ classId;
        if( in_groupcount[ index ] > 0 )
        {

            addToGroup( out_groups, groupIndex, temp );
            in_groupcount[ index ]--;
        }
        temp = temp->next;
    }
}



void normalizeOpcodeFrequency( 
        s_files ** out_filelist
        ) 
{
    s_fileProp * temp;
    temp = (*out_filelist)->list;
    int numfiles = (*out_filelist)->numFiles;
    int i=0, j=0;
    int min=0, max=0;
    for ( i=0; i<numfiles; i++)
    {
        min = temp->min_opcodefreq;
        max = temp->max_opcodefreq;
        s_opcodenode *opc = temp->opcodes; 
        float *nopc = temp->normalized_opcodes;
        for ( j=0; j<temp->numopcode; j++)
        {
            nopc[j] = (float)(opc[j].freq - min)/(float)(max - min);
            assert( nopc[j] >= 0.0f ); // TODO can decide a threshold, right now checks only for +ve values
        }
        temp=temp->next;
    }
}

int fillGroupWiseData(
        s_group   *in_groups,
        float     *out_trainArray,
        int       in_num_groups,
        int       in_num_opcodes,
        float     *out_testArray,
        int       *out_class_vect,
        int       *out_group_vect
        )
{
    int mean =0, var =1;
    int i,j,k,opcindex;
    float x;
    int fcount=0;
    int numtestfiles=0;

    for ( i=0; i<in_num_groups; i++)
    {
        for ( j=0; j<2; j++)
        {
            s_filelistnode *file = in_groups[i].list[j];
            int numfiles = in_groups[i].count*2;
            if( numfiles > 0)
            {
                while( file != NULL )
                {
                    fcount++;
                    if ( fcount%3 != 0)
                    {
                        for ( k=0; k< file->prop->numopcode; k++)
                        {
                            //printf(" %f", file->prop->normalized_opcodes[k]);
                            x = file->prop->normalized_opcodes[k];
                            opcindex = file->prop->opcodes[k].id;
                            float m = x/numfiles;
                            float v = (m-x)*(m-x)/numfiles;
                            int row = i*4;
                            int cls = file->prop->classId*2;
                            assert( m >= 0.0f && v >= 0.0f);
                            out_trainArray[((row+cls+mean)*in_num_opcodes)+opcindex] += m;
                            out_trainArray[((row+cls+var )*in_num_opcodes)+opcindex] += v;
                        }
                    }
                    else // fcount%3 == 0
                    {
                        fcount = 0;
                        for ( k=0; k< file->prop->numopcode; k++)
                        {
                            opcindex = file->prop->opcodes[k].id;
                            x = file->prop->normalized_opcodes[k];
                            out_testArray[ (numtestfiles*in_num_opcodes) + opcindex] += x;
                            out_class_vect[numtestfiles] = file->prop->classId;
                            out_group_vect[numtestfiles] = (file->prop->size)/5;
                        }
                        numtestfiles++;
                    }
                    file = file->next;
                }
            }
        }
    }
    return numtestfiles;
}


void resetVector( 
        float * out_vector, 
        int in_num_columns
        )
{
    int i;
    for ( i=0; i<in_num_columns; i++)
    {
        out_vector[i] = 0.0;
    }
}

void selectFeaturesForEachGroup(
        s_group ** out_group,
        int in_num_groups,
        int in_num_opcodes,
        int in_num_features 
        )
{
    int i,j,k,l;
    float **features=(float**) calloc( sizeof( float* ) , 2 );
    features[0]=(float*) calloc( sizeof( float ) , in_num_opcodes );
    features[1]=(float*) calloc( sizeof( float ) , in_num_opcodes );

    s_group *grp_ptr;
    for ( i=0; i<in_num_groups; i++)
    {
        grp_ptr = &((*out_group)[i]);
        if( grp_ptr->count > 0 )
        {
            resetVector( features[0], in_num_opcodes);
            resetVector( features[1], in_num_opcodes);

            for ( j=0; j< grp_ptr->count; j++)
            {
                for ( k=0; k<2; k++) // TODO NUM_CLASSES
                {
                    s_filelistnode *file = grp_ptr->list[k];
                    while( file != NULL )
                    {
                        s_fileProp *fileprop_ptr = file->prop;
                        for ( l=0; l<file->prop->numopcode; l++)
                        {
                            int opcindex = fileprop_ptr->opcodes[l].id;
                            int freq = fileprop_ptr->opcodes[l].freq;
                            features[k][opcindex] += (freq/grp_ptr->count) ; // TODO divided by number of files in each class
                        }
                        file=file->next;
                    }
                }

            }
            setFeatureVector( features, grp_ptr, 2, in_num_opcodes, in_num_features); // TODO NUM_CLASSES
        }
    }
    free( features[0] );
    free( features[1] );
    free( features );
}

int cmpopcodenode( 
        const void * opc1, 
        const void *opc2
        )
{
    s_diffnode a = *(s_diffnode const*) opc1;
    s_diffnode b = *(s_diffnode const*) opc2;

    if( a.diff < b.diff ) return 1; /// sorts in ascending order
    else return 0;
}

void setFeatureVector( 
        float **in_features, 
        s_group * out_group , 
        int in_num_list, 
        int in_num_columns, 
        int in_num_features 
        )
{
    s_diffnode * diffvector = (s_diffnode*) calloc( sizeof(s_diffnode), in_num_columns);
    int j=0;
    for ( j=0; j<in_num_columns; j++)
    {
        diffvector[j].id = j;
        diffvector[j].diff = abs( in_features[0][j] - in_features[1][j] ); // TODO NUM_CLASSES
    }

    /// sort in ascending order
    qsort( diffvector, in_num_columns, sizeof(s_opcodenode), cmpopcodenode);

    out_group->features = (int*) calloc( sizeof(int), in_num_columns);

    for ( j=0; j<in_num_features; j++)
    {
        int opcindex = diffvector[j].id;
        out_group->features[ opcindex ] = 1;
        //printf(" %d", opcindex);
    }
    //printf(" \n");
    free( diffvector );
}

void assignFeatureListForEachGroup( 
        int ***out_feature_list, 
        s_group *in_groups, 
        int in_num_groups
        )
{
    int i;
    for ( i=0; i<in_num_groups; i++)
    {
        if( in_groups[i].count > 0 )
            (*out_feature_list)[i] = in_groups[i].features;
            else
                (*out_feature_list)[i] = NULL;
    }
}
void spillFeatureMatrix(
    int **in_featureptr,
    int *out_featurematrix,
    int in_numgroups,
    int in_numopcode
    )
{
    int i,j;
    for( i=0; i<in_numgroups; i++)
    {
        for( j=0; j<in_numopcode; j++)
        {
            if( in_featureptr[i] )
                out_featurematrix[ i*in_numopcode+j ] = in_featureptr[i][j];
            else
                out_featurematrix[ i*in_numopcode+j ] = 0;
        }
    }
    }

void showFiles( 
        s_files * p_files 
        )
{
    s_fileProp *temp = p_files->list;
    s_opcodenode *opcptr = NULL;
    int count = p_files->numFiles;
    while ( temp != NULL)
    {
        printf("%d %s %d\n", count--, temp->name, temp->numopcode);
        opcptr = temp->opcodes;
        int i=0;
        while( i<temp->numopcode)
        {
            printf( " %d=%d\t", opcptr[i].id, opcptr[i].freq);
            i++;
        }
        temp = temp->next;
    }
}

void showGroupWiseStats( 
        s_group * in_groups , 
        int in_num_groups
        )
{
    int i,j,c;
    for( i=0; i< in_num_groups; i++)
    {
        printf(" Group %d, has %d files, max opcode count = %d, min opcode count = %d.\n", \
                i+1, in_groups[i].count, in_groups[i].max, in_groups[i].min);
        s_filelistnode *temp;
        for( c=0 ; c<2; c++) // TODO make this genric NUM_CLASSES
        {
            temp = in_groups[i].list[c];
            for ( j=0; j<in_groups[i].count; j++)
            {
                printf(" %d ",temp->prop->numopcode); 
                temp = temp->next;
            }
            printf("\n");
        }
    }
}

void showGroupWiseProcessedValues( 
        float *out_data_matrix, 
        int in_num_groups, 
        int in_num_opcodes
        )
{
    int mean =0, var =1;
    int i,j;
    FILE *fp;
    char *fname = "/tmp/file";
    fp = fopen(fname, "w");

    printf(" Writing data to file %s\n", fname);
    for ( i=0; i<in_num_groups*4; i+=4)
    {
        for ( j=0; j<in_num_opcodes; j++)
        {
            fprintf(fp," %f", out_data_matrix[((i+0+mean)*in_num_opcodes)+j]);
        }
        fprintf(fp,"\n");
        for ( j=0; j<in_num_opcodes; j++)
        {
            fprintf(fp," %f", out_data_matrix[((i+0+var)*in_num_opcodes)+j]);
        } 
        fprintf(fp,"\n");
        for ( j=0; j<in_num_opcodes; j++)
        {
            fprintf(fp," %f", out_data_matrix[((i+2+mean)*in_num_opcodes)+j]);
        }
        fprintf(fp,"\n");
        for ( j=0; j<in_num_opcodes; j++)
        {
            fprintf(fp," %f", out_data_matrix[((i+2+var)*in_num_opcodes)+j]);
        } 
        fprintf(fp,"\n");
    }

    fclose(fp);
}

void spillMatrixToFile( float *in_mat, int in_numrows, int in_numcolumns, char *filename)
{
    FILE *fp ;
    fp = fopen(filename, "w");

    int i,j;
    for ( i=0; i<in_numrows; i++)
    {
        for ( j=0; j<in_numcolumns; j++)
        {
            fprintf(fp,"%0.5f ",in_mat[ i*in_numcolumns + j ] );
            }
            fprintf(fp,"\n");
    }
    fclose( fp );
}
