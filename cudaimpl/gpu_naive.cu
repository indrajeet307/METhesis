#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "gpu_naive.h"
#define BLOCK_WIDTH 64
hipError_t err;
hipEvent_t starttimer, endtimer;
/*!
*	\brief Allocates rows X columns size of float matrix on device
*
*   Need to pass address of the pointer
*	\return  
*	
*/
void createDeviceMatrixF(
        float **mat,    /*! [out] Matrix pointer on device  */
        int rows,       /*!< [in] number of rows in the matrix */
        int columns     /*!< [in] number of columns in the matrix */
        )
{
    err = hipSuccess;
    err = hipMalloc( mat, rows*columns*sizeof(float) );
    if ( err != hipSuccess )
    {
        fprintf(stderr, "#Error %s, %d.\n%s.", __FILE__, __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    // set all the values to zero 
    err = hipMemset((*mat), 0, rows*columns*sizeof(float));
    if ( err != hipSuccess )
    {
        fprintf(stderr, "#Error %s, %d.\n%s.", __FILE__, __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

/*!
*	\brief Allocates rows X columns size of int matrix on device
*
*   Need to pass address of the pointer
*	\return  
*	
*/
void createDeviceMatrixI(
        int **mat,     /*! [out] Matrix pointer on device  */
        int rows,      /*!< [in] number of rows in the matrix */
        int columns    /*!< [in] number of columns in the matrix */
        )
{
    err = hipSuccess;
    err = hipMalloc( mat, rows*columns*sizeof(int) );
    if ( err != hipSuccess )
    {
        fprintf(stderr, "#Error %s, %d.\n%s.", __FILE__, __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    // set all the values to zero 
    err = hipMemset((*mat), 0, rows*columns*sizeof(int));
    if ( err != hipSuccess )
    {
        fprintf(stderr, "#Error %s, %d.\n%s.", __FILE__, __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

/*!
*	\brief Transfer an Integer vector of size from host to the device memory
*
*   More Details ...
*	\return  
*	
*/
void transferToDeviceI(
        int *hostptr,     /*!< [in] host memory pointer */
        int *deviceptr,     /*!< [in] device memory pointer */
        int size    /*!< [in] size of data to be transfered in bytes */
        )
{
    err = hipSuccess;
    err = hipMemcpy(deviceptr, hostptr, sizeof(int)*size, hipMemcpyHostToDevice);
    if(err != hipSuccess)
    {
        fprintf(stderr, "#Error %s, %d.\n%s.", __FILE__, __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

/*!
*	\brief Transfer an Float vector of size from host to the device memory
*
*   More Details ...
*	\return  
*	
*/
void transferToDeviceF(
        float *hostptr,     /*!< [in] host memory pointer */
        float *deviceptr,     /*!< [in] device memory pointer */
        int size    /*!< [in] size of data to be transfered in bytes */
        )
{
    err = hipSuccess;
    err = hipMemcpy(deviceptr, hostptr, sizeof(float)*size, hipMemcpyHostToDevice);
    if(err != hipSuccess)
    {
        fprintf(stderr, "#Error %s, %d.\n%s.", __FILE__, __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

/*!
*	\brief Transfer an Integer vector of size from device to the host memory
*
*   More Details ...
*	\return  
*	
*/
void transferFromDeviceI(
        int *hostptr,     /*!< [in] host memory pointer */
        int *deviceptr,     /*!< [in] device memory pointer */
        int size    /*!< [in] size of data to be transfered in bytes */
        )
{
    err = hipSuccess;
    err = hipMemcpy(hostptr, deviceptr, sizeof(int)*size, hipMemcpyDeviceToHost);
    if(err != hipSuccess)
    {
        fprintf(stderr, "#Error %s, %d.\n%s.", __FILE__, __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

/*!
*	\brief Transfer an Integer vector of size from device to the host memory
*
*   More Details ...
*	\return  
*	
*/
void transferFromDeviceF(
        float *hostptr,     /*!< [in] host memory pointer */
        float *deviceptr,     /*!< [in] device memory pointer */
        int size    /*!< [in] size of data to be transfered in bytes */
        )
{
    err = hipSuccess;
    err = hipMemcpy(hostptr, deviceptr, sizeof(float)*size, hipMemcpyDeviceToHost);
    if(err != hipSuccess)
    {
        fprintf(stderr, "#Error %s, %d.\n%s.", __FILE__, __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

/*!
*	\brief Creates the probability matrix on the device from the input matrix
*
*   Takes input a matrix containing normalized opcode frequency values, genrates one
*   matrix and a vector, matrix is num_classes X num_opcodes wide and has probability of
*   opcode for that class, the vector contains ratio of number of files of a class to
*   total number of files
*
*   inmat: input matrix, opcodes(along y) X num_files(along x)
*
*   inclass: vector, containg class to which the current file belongs
*
*   class_wise: total opcodes in each class
*
*   outmat: matrix is num_classes X num_opcodes wide and has probability of
*   opcode for that class
*
*   class_prob: the vector contains ratio of number of files of a class to
*   total number of files
*
*   inrows: total number of opcodes
*   incolumns: total number of files for training 
*   outrows: total number of classes 
*
*	\return 
*	
*/
__global__
void naiveTrainKernel(
        int *inmat,     /*!< [in] input matrix */
        int *inclass,     /*!< [in] input class, for each file */
        int *class_wise,    /*!< [out] class wise total opcodes */
        float *outmat,     /*!< [out] probablity of each opcode in each class */
        float *class_prob,     /*!< [out] probility of each class */
        int inrows,     /*!< [in] number of opcodes*/
        int incolumns,     /*!< [in] total number of files*/
        int outrows    /*!< [in] number of classes */
        )
{
    // thread for each file
    int index = blockIdx.x*BLOCK_WIDTH+threadIdx.x;
    int cls;
    if( index < incolumns)
    {
        cls = inclass[ index ];

        // for each opcode
        for ( int i=0; i<inrows; i++)
        {
            // increment the count of total opcodes in current class by the frequency in
            // input matrix
            atomicAdd( &(class_wise[ cls ]), inmat[ i*incolumns + index]);
            // increment the count of opcode in current class by the frequency in input 
            // matrix
            atomicAdd( &(outmat[ cls*inrows + i]), (float)inmat[ i*incolumns + index]);
            // increment the number of files in current class
            atomicAdd( &(class_prob[ cls ]), (float)1);
        }
        __syncthreads();
        for ( int i=0; i<inrows; i++)
        {
            // save the probability of each opcode
            float temp = ( log10(( outmat[ cls*inrows+ i]+1)/ (class_wise[ cls ]+1)));
            outmat[ cls*inrows+ i] = (-1)*temp;
        }
        __syncthreads();
        // save the probability of each class 
        class_prob[ cls ] = (-1)*log10( class_prob[ cls ]/incolumns );
    }
}

/*!
*	\brief Wrapper funtion for traning the kernel
*
*   For variable description refer the kernel documentation
*	\return  
*	
*/
void pnaiveTrain(
        int *inmat,     /*!< [in] input matrix */
        int *inclass,     /*!< [in] input class, for each file */
        int *class_wise,    /*!< [out] class wise total opcodes */
        float *outmat,     /*!< [out] probablity of each opcode in each class */
        float *class_prob,     /*!< [out] probility of each class */
        int inrows,     /*!< [in] number of opcodes*/
        int incolumns,     /*!< [in] total number of files*/
        int outrows    /*!< [in] number of classes */
        )
{
    dim3 gridProp( ceil(incolumns/BLOCK_WIDTH)+1,1,1);
    dim3 blockProp(BLOCK_WIDTH,1,1);
    printf(" Running %d Threads.\n",BLOCK_WIDTH);
    printf(" Running %.0lf Blocks.\n",ceil(incolumns/BLOCK_WIDTH));
    err = hipSuccess;
    naiveTrainKernel<<<gridProp,blockProp>>>( inmat, inclass, class_wise, outmat, class_prob, inrows, incolumns, outrows);
    if (err != hipSuccess)
    {
        fprintf(stderr, "%s, %d.\n %s.", __FILE__, __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

/*!
*	\brief Assigns classes to the unassigned files, using the probability matrix genrated
*	by the training kernel
*
*   in_probmat : probability matrix, genrated by training kernel
*
*   in_class_prob : vector containing, ratio of number of files in a class to total number
*   of files
*
*   in_test_mat : opcode frequency matrix for test file
*
*   in_test_mat_columns : number of files for testing
*
*   in_test_mat_rows : total number of opcodes
*
*   in_probmat_rows : number of classes 
*
*   in_probmat_columns : total number of opcodes 
*
*   out_assigned_class : predicted class for the test files
*   
*	\return 
*	
*/
__global__
void naiveTestKernel(
        float *in_probmat,          /*!< [in] pobability of each opcode in each class */
        float *in_class_prob,       /*!< [in] probability of each class */
        int *in_test_mat,           /*!< [in] input test files */
        int  in_test_mat_columns,   /*!< [in] number of files in test matrix */
        int  in_test_mat_rows,      /*!< [in] number of opcodes in test matrix */
        int  in_probmat_rows,       /*!< [in] number of classes */
        int  in_probmat_columns,    /*!< [in] number of opcodes */
        int *out_assigned_class     /*!< [out] class assigned to input files */
        )
{
    int index = blockIdx.x*BLOCK_WIDTH+threadIdx.x;
    float cls[2];
    if( index < in_test_mat_columns)
    {
        cls[0] = cls[1] = 0;
        // for each opcode
        for( int i =0 ;i<in_test_mat_rows;i++)
        {
            // if the opcode is present in the test file
            if( in_test_mat[ i*in_test_mat_columns+index ] > 0)
            {
                // for each class
                for ( int j=0; j<in_probmat_rows; j++)
                {
                    // update the probability 
                    cls[j] += (float)in_test_mat[ i*in_test_mat_columns+index]*in_probmat[ j*in_probmat_columns+ i];
                }
            }
        }
        // assign the class with maximum probability
        out_assigned_class[ index ] = cls[0] > cls[1] ? 0 : 1;
    }
}

/*!
*	\brief Wrapper funtion for testing the kernel genrated by training
*
*   For variable description refer the kernel documentation
*	\return  
*	
*/
void pnaiveTest(
        float *in_probmat,          /*!< [in] pobability of each opcode in each class */
        float *in_class_prob,       /*!< [in] probability of each class */
        int *in_test_mat,           /*!< [in] input test files */
        int  in_test_mat_columns,   /*!< [in] number of files in test matrix */
        int  in_test_mat_rows,      /*!< [in] number of opcodes in test matrix */
        int  in_probmat_rows,       /*!< [in] number of classes */
        int  in_probmat_columns,    /*!< [in] number of opcodes */
        int *out_assigned_class     /*!< [out] class assigned to input files */
        )
{
    dim3 gridProp( ceil(in_test_mat_columns/BLOCK_WIDTH)+1,1,1);
    dim3 blockProp(BLOCK_WIDTH,1,1);
    printf(" Running %d Threads.\n",BLOCK_WIDTH);
    printf(" Running %.0lf Blocks.\n",ceil(in_test_mat_columns/BLOCK_WIDTH));
    err = hipSuccess;
    naiveTestKernel<<<gridProp,blockProp>>>(in_probmat, in_class_prob, in_test_mat,
            in_test_mat_columns, in_test_mat_rows, in_probmat_rows, 
            in_probmat_columns, out_assigned_class);
    if (err != hipSuccess)
    {
        fprintf(stderr, "%s, %d.\n %s.", __FILE__, __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

/*!
 *	\brief Gives the probablity for the current in_val
 *
 *   probability = \f$ \frac{1}{\sqrt{2\Pi\sigma^{2}}}\exp^{\frac{(x-\mu)^{2}}{2\sigma^{2}}} \f$
 *
 *	\return probablity in float
 *	\see 
 *	
 */
__device__
float getTheProbablityD( 
        float in_vval,  /*!< [in] x as in above formulae */
        float in_vmean, /*!< [in] mean value */
        float in_vvar   /*!< [in] variance value */
        )
{

    float result=0.0;
    float val1 =  1/sqrt( 2.0* M_PI* in_vvar);
    float val2 = (in_vval-in_vmean)*(in_vval-in_vmean)/(2.0*in_vvar);
    val2 = 1 / exp( val2);
    result = log10( val1*val2);
    if( isnan(result) || isinf(result) ) return 0.0;
    return result;
}

/*!
*	\brief Assigns class based on mean and variance of the frequency of opcode occurance
*	in the file
*
*   Threads equal to number of test files are created, each calculates the probability of
*   current file being benign or malware, the class with maximum probability is assigned
*   Each thread goes through each opcode if the normalized value of current opcode is
*   greater than 0, calulate and add the probability
*
*   in_trainedMatrix : matrix containing the groupwise mean and variance values
*
*   in_testMatrix : test files normalized opcode frequency values 
        numtesfiles X numopcodes
*
*   in_numgroups : total number of groups    
*
*   in_numopcode : total number of opcodes    
*
*   in_numtestfiles : number of test files  
*
*   in_groupindexvector : group index of the current file
*
*   out_predictvector : vector of predicted class
*
*	\return What does it return?
*	
*/
__global__
void assignClassUsingMeanVarianceDataKernel(
        float *in_trainedMatrix,    /*!< [in] trained Matrix, containing mean variance
        values */
        float *in_testMatrix,    /*!< [in] test files */
        int in_numgroups,    /*!< [in] number of groups */
        int in_numopcode,    /*!< [in] number of opcodes */
        int in_numtestfiles,    /*!< [in] number of test files */
        int *in_groupindexvector,    /*!< [in] vector containing group index of test files
        based on its size*/
        int *out_predictvector    /*!< [out] predicted class for each input file */
        )
{
    int tid = threadIdx.x + blockIdx.x * BLOCK_WIDTH;
    if( tid < in_numtestfiles )
    {

        int group_index = in_groupindexvector[tid];
        int index_in_trainedMatrix = group_index*4;
        int im=0, iv=1;
        float bprob=0.0, mprob=0.0;

        __syncthreads();

        // for each opcode
        for( int i=0; i<in_numopcode; i++)
        {
            float x;
            x = in_testMatrix[ i*in_numtestfiles + tid ];
            // if the test file has the current opcode
            if( x > 0) 
            {
                float bvar, bmean, mvar, mmean;
                bmean = in_trainedMatrix[ (0+im+index_in_trainedMatrix)*in_numopcode + i];
                bvar  = in_trainedMatrix[ (0+iv+index_in_trainedMatrix)*in_numopcode + i];
                // get probability of file being benign
                bprob += getTheProbablityD( x, bmean, bvar);

                mmean = in_trainedMatrix[ (2+im+index_in_trainedMatrix)*in_numopcode+ i];
                mvar  = in_trainedMatrix[ (2+iv+index_in_trainedMatrix)*in_numopcode+ i];
                // get probability of file being malware
                mprob += getTheProbablityD( x, mmean, mvar);
            }

        }
        __syncthreads();

        // assign class with maximum probability
        if( bprob > mprob) 
            out_predictvector[ tid ] = 0;
        else
            out_predictvector[ tid ] = 1;
    }
    __syncthreads();
}

/*!
*	\brief Wrapper funtion for testing using the mean and variance data 
*
*   For variable description refer the kernel documentation
*	\return  
*	
*/
void passignClassUsingMeanVarianceData( 
        float *in_trainedMatrix,    /*!< [in] trained Matrix, containing mean variance
        values */
        float *in_testMatrix,    /*!< [in] test files */
        int in_numgroups,    /*!< [in] number of groups */
        int in_numopcode,    /*!< [in] number of opcodes */
        int in_numtestfiles,    /*!< [in] number of test files */
        int *in_groupindexvector,    /*!< [in] vector containing group index of test files
        based on its size*/
        int *out_predictvector    /*!< [out] predicted class for each input file */
        )
{
    /*
TODO: try checking if we can get speed up by making trainedMatrix go to 
constant memory
     */
    dim3 gridProp( ceil(in_numtestfiles/BLOCK_WIDTH)+1,1,1);
    dim3 blockProp(BLOCK_WIDTH,1,1);
    printf(" Running Kernel %d Threads.\n",BLOCK_WIDTH);
    printf(" Running %.0lf Blocks.\n",ceil(in_numtestfiles/BLOCK_WIDTH));
    err = hipSuccess;
    startCudaTimer();
    assignClassUsingMeanVarianceDataKernel<<<gridProp,blockProp>>>(
            in_trainedMatrix,
            in_testMatrix,
            in_numgroups,
            in_numopcode,
            in_numtestfiles,
            in_groupindexvector,
            out_predictvector
            );
    endCudaTimer();
    printf(" Time required for parallel is %f\n",getCudaTime());
    if (err != hipSuccess)
    {
        fprintf(stderr, "%s, %d.\n %s.", __FILE__, __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

}

/*!
*	\brief Assigns class based on mean and variance of the frequency of opcode occurance
*	in the file, only opcodes belonging to the prominent group (feature list) are taken into
*	consideration
*
*   Threads equal to number of test files are created, each calculates the probability of
*   current file being benign or malware, the class with maximum probability is assigned
*   Each thread goes through each opcode if the normalized value of current opcode is
*   greater than 0 and the opcode belongs to the prominent opcode( freature list), 
*   calulate and add the probability
*
*   in_trainedMatrix : matrix containing the groupwise mean and variance values
*
*   in_testMatrix : test files normalized opcode frequency values 
*        numtesfiles X numopcodes
*
*   in_featureMatrix : (promenent opcodes) features for each group
*
*   in_numgroups : total number of groups    
*
*   in_numopcode : total number of opcodes    
*
*   in_numtestfiles : number of test files  
*
*   in_groupindexvector : group index of the current file
*
*   out_predictvector : vector of predicted class
*
*	\return What does it return?
*	
*/
    __global__
void assignClassUsingMeanVarianceDataUsingFeatureSelectionKernel(
        float *in_trainedMatrix,    /*!< [in] trained Matrix, containing mean variance
        values */
        float *in_testMatrix,    /*!< [in] test files */
        int *in_featureMatrix,       /*!< [in] feature vector, for each group*/
        int in_numgroups,    /*!< [in] number of groups */
        int in_numopcode,    /*!< [in] number of opcodes */
        int in_numtestfiles,    /*!< [in] number of test files */
        int *in_groupindexvector,    /*!< [in] vector containing group index of test files
        based on its size*/
        int *out_predictvector    /*!< [out] predicted class for each input file */
        )
{
    int tid = threadIdx.x + blockIdx.x * BLOCK_WIDTH;
    int group_index = in_groupindexvector[tid];
    int index_in_trainedMatrix = group_index*4;
    int im=0, iv=1;
    float bprob=0.0, mprob=0.0;
    __syncthreads();

    if( tid < in_numtestfiles)
    {
    // for each opcode
    for( int i=0; i<in_numopcode; i++)
    {
        int feature;
        float x;
        x = in_testMatrix[ i*in_numtestfiles + tid ];
        feature = in_featureMatrix[ group_index*in_numopcode+ i];
        // if opcode present in test file
        if( x > 0) 
        {
            float bvar=0, bmean=0, mvar=0, mmean=0;
            bmean = in_trainedMatrix[ (0+im+index_in_trainedMatrix)*in_numopcode + i];
            bvar  = in_trainedMatrix[ (0+iv+index_in_trainedMatrix)*in_numopcode + i];
            // get the probability of file being benign
            bprob += getTheProbablityD( x, bmean, bvar)*feature;
            // multiplied with feature(0/1), will account only if the feature is amongst
            // the selected feature

            mmean = in_trainedMatrix[ (2+im+index_in_trainedMatrix)*in_numopcode+ i];
            mvar  = in_trainedMatrix[ (2+iv+index_in_trainedMatrix)*in_numopcode+ i];
            // get the probability of file being malware 
            mprob += getTheProbablityD( x, mmean, mvar)*feature;
        }
    }
    __syncthreads();

    // assign the class
    if( bprob > mprob ) 
        out_predictvector[ tid ] = 0;
    else
        out_predictvector[ tid ] = 1;
    }
}

/*!
*	\brief Wrapper funtion for testing using the mean and variance data and feature
*	selection vector
*
*   For variable description refer the kernel documentation
*	\return  
*	
*/
void passignClassUsingMeanVarianceDataUsingFeatureSelection( 
        float *in_trainedMatrix,    /*!< [in] trained Matrix, containing mean variance
        values */
        float *in_testMatrix,    /*!< [in] test files */
        int *in_featureMatrix,       /*!< [in] feature vector */
        int in_numgroups,    /*!< [in] number of groups */
        int in_numopcode,    /*!< [in] number of opcodes */
        int in_numtestfiles,    /*!< [in] number of test files */
        int *in_groupindexvector,    /*!< [in] vector containing group index of test files
        based on its size*/
        int *out_predictvector    /*!< [out] predicted class for each input file */
        )
{
    /*
TODO: try checking if we can get speed up by making trainedMatrix go to 
constant memory
     */
    dim3 gridProp( ceil(in_numtestfiles/BLOCK_WIDTH)+1,1,1);
    dim3 blockProp(BLOCK_WIDTH,1,1);
    printf(" Running Kernel %d Threads.\n",BLOCK_WIDTH);
    printf(" Running %.0lf Blocks.\n",ceil(in_numtestfiles/BLOCK_WIDTH));
    err = hipSuccess;
    startCudaTimer();
    assignClassUsingMeanVarianceDataUsingFeatureSelectionKernel<<<gridProp,blockProp>>>(
            in_trainedMatrix,
            in_testMatrix,
            in_featureMatrix,
            in_numgroups,
            in_numopcode,
            in_numtestfiles,
            in_groupindexvector,
            out_predictvector
            );
    if (err != hipSuccess)
    {
        fprintf(stderr, "%s, %d.\n %s.", __FILE__, __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    endCudaTimer();
    printf(" Time required for parallel is %f\n",getCudaTime());
}

void startCudaTimer()
{
    hipEventCreate(&starttimer);
    hipEventCreate(&endtimer);
    hipEventRecord(starttimer);

}
void endCudaTimer()
{
    hipEventRecord(endtimer);
}
float getCudaTime()
{
    float time;
    hipEventSynchronize(endtimer);
    hipDeviceSynchronize();
    hipEventElapsedTime(&time, starttimer,endtimer);
    return time;
}
