#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "gpu_naive.h"
#define BLOCK_WIDTH 64
hipError_t err;
void createDeviceMatrixF(float **mat, int rows, int columns)
{
     err = hipSuccess;
     err = hipMalloc( mat, rows*columns*sizeof(float) );
     if ( err != hipSuccess )
     {
          fprintf(stderr, "#Error %s, %d.\n%s.", __FILE__, __LINE__, hipGetErrorString(err));
          exit(EXIT_FAILURE);
     }
     err = hipMemset((*mat), 0, rows*columns*sizeof(float));
     if ( err != hipSuccess )
     {
          fprintf(stderr, "#Error %s, %d.\n%s.", __FILE__, __LINE__, hipGetErrorString(err));
          exit(EXIT_FAILURE);
     }
}

void createDeviceMatrixI(int **mat, int rows, int columns)
{
     err = hipSuccess;
     err = hipMalloc( mat, rows*columns*sizeof(int) );
     if ( err != hipSuccess )
     {
          fprintf(stderr, "#Error %s, %d.\n%s.", __FILE__, __LINE__, hipGetErrorString(err));
          exit(EXIT_FAILURE);
     }
     err = hipMemset((*mat), 0, rows*columns*sizeof(int));
     if ( err != hipSuccess )
     {
          fprintf(stderr, "#Error %s, %d.\n%s.", __FILE__, __LINE__, hipGetErrorString(err));
          exit(EXIT_FAILURE);
     }
}

void transferToDeviceI(int *hostptr, int *deviceptr, int size)
{
     err = hipSuccess;
     err = hipMemcpy(deviceptr, hostptr, sizeof(float)*size, hipMemcpyHostToDevice);
     if(err != hipSuccess)
     {
          fprintf(stderr, "#Error %s, %d.\n%s.", __FILE__, __LINE__, hipGetErrorString(err));
          exit(EXIT_FAILURE);
     }
}

void transferToDeviceF(float *hostptr, float *deviceptr, int size)
{
     err = hipSuccess;
     err = hipMemcpy(deviceptr, hostptr, sizeof(float)*size, hipMemcpyHostToDevice);
     if(err != hipSuccess)
     {
          fprintf(stderr, "#Error %s, %d.\n%s.", __FILE__, __LINE__, hipGetErrorString(err));
          exit(EXIT_FAILURE);
     }
}

void transferFromDeviceI(int *hostptr, int *deviceptr, int size)
{
     err = hipSuccess;
     err = hipMemcpy(hostptr, deviceptr, sizeof(int)*size, hipMemcpyDeviceToHost);
     if(err != hipSuccess)
     {
          fprintf(stderr, "#Error %s, %d.\n%s.", __FILE__, __LINE__, hipGetErrorString(err));
          exit(EXIT_FAILURE);
     }
}

void transferFromDeviceF(float *hostptr, float *deviceptr, int size)
{
     err = hipSuccess;
     err = hipMemcpy(hostptr, deviceptr, sizeof(float)*size, hipMemcpyDeviceToHost);
     if(err != hipSuccess)
     {
          fprintf(stderr, "#Error %s, %d.\n%s.", __FILE__, __LINE__, hipGetErrorString(err));
          exit(EXIT_FAILURE);
     }
}

/*
   files along x and opcodes along y
 */
     __global__
void naiveTrainKernel(int *inmat, int *inclass, int *class_wise,float *outmat, float *class_prob, int inrows, int incolumns, int outrows)
{
     int index = blockIdx.x*BLOCK_WIDTH+threadIdx.x;
     int cls;
     if( index < incolumns)
     {
          cls = inclass[ index ];

          for ( int i=0; i<inrows; i++)
          {
               atomicAdd( &(class_wise[ cls ]), inmat[ i*incolumns + index]);
               atomicAdd( &(outmat[ cls*inrows + i]), (float)inmat[ i*incolumns + index]);
               atomicAdd( &(class_prob[ cls ]), (float)1);
          }
          __syncthreads();
          for ( int i=0; i<inrows; i++)
          {
               float temp = ( log10(( outmat[ cls*inrows+ i]+1)/ (class_wise[ cls ]+1)));
               outmat[ cls*inrows+ i] = (-1)*temp;
          }
          __syncthreads();
          class_prob[ cls ] = (-1)*log10( class_prob[ cls ]/incolumns );
     }
}

void pnaiveTrain( int *inmat, int *inclass, int *class_wise, float *outmat, float *class_prob, int inrows, int incolumns, int outrows)
{
     dim3 gridProp( ceil(incolumns/BLOCK_WIDTH)+1,1,1);
     dim3 blockProp(BLOCK_WIDTH,1,1);
     printf(" Running %d Threads.\n",BLOCK_WIDTH);
     printf(" Running %.0lf Blocks.\n",ceil(incolumns/BLOCK_WIDTH));
     err = hipSuccess;
     naiveTrainKernel<<<gridProp,blockProp>>>( inmat, inclass, class_wise, outmat, class_prob, inrows, incolumns, outrows);
     if (err != hipSuccess)
     {
          fprintf(stderr, "%s, %d.\n %s.", __FILE__, __LINE__, hipGetErrorString(err));
          exit(EXIT_FAILURE);
     }
}

__global__

void naiveTestKernel( float *in_probmat, float *in_class_prob, int*in_test_mat,
          int in_test_mat_columns, int in_test_mat_rows, int in_probmat_rows, 
          int in_probmat_columns, int *out_assigned_class)
{
     int index = blockIdx.x*BLOCK_WIDTH+threadIdx.x;
     float cls[2];
     if( index < in_test_mat_columns)
     {
          cls[0] = cls[1] =0;
          for( int i =0 ;i<in_test_mat_rows;i++)
          {
               if( in_test_mat[ i*in_test_mat_columns+index ] > 0)
               {
                    for ( int j=0; j<in_probmat_rows; j++)
                    {
                         cls[j] += (float)in_test_mat[ i*in_test_mat_columns+index]*in_probmat[
                              j*in_probmat_columns+ i];
                    }
               }
          }
          out_assigned_class[ index ] = cls[0] > cls[1] ? 0 : 1;
     }
}

void pnaiveTest( float *in_probmat, float *in_class_prob, int*in_test_mat,
          int in_test_mat_columns, int in_test_mat_rows, int in_probmat_rows, 
          int in_probmat_columns, int *out_assigned_class)
{
     dim3 gridProp( ceil(in_test_mat_columns/BLOCK_WIDTH)+1,1,1);
     dim3 blockProp(BLOCK_WIDTH,1,1);
     printf(" Running %d Threads.\n",BLOCK_WIDTH);
     printf(" Running %.0lf Blocks.\n",ceil(in_test_mat_columns/BLOCK_WIDTH));
     err = hipSuccess;
     naiveTestKernel<<<gridProp,blockProp>>>(in_probmat, in_class_prob, in_test_mat,
               in_test_mat_columns, in_test_mat_rows, in_probmat_rows, 
               in_probmat_columns, out_assigned_class);
     if (err != hipSuccess)
     {
          fprintf(stderr, "%s, %d.\n %s.", __FILE__, __LINE__, hipGetErrorString(err));
          exit(EXIT_FAILURE);
     }
}

__device__
float getTheProbablityD( 
          float in_vval,  /*!< [in] x as in above formulae */
          float in_vmean, /*!< [in] mean value */
          float in_vvar   /*!< [in] variance value */
          )
{

     float result=0.0;
     float val1 =  1/sqrt( 2.0* M_PI* in_vvar);
     float val2 = (in_vval-in_vmean)*(in_vval-in_vmean)/(2.0*in_vvar);
     val2 = 1 / exp( val2);
     result = log10( val1*val2);
     if( isnan(result) || isinf(result) ) return 0.0;
     return result;
}
     __global__
void assignClassUsingMeanVarianceDataKernel(
          float *in_trainedMatrix,
          float *in_testMatrix,
          int in_numgroups,
          int in_numopcode,
          int in_numtestfiles,
          int *in_groupindexvector,
          int *out_predictvector
          )
{
     int tid = threadIdx.x + blockIdx.x * BLOCK_WIDTH;
     if( tid < in_numtestfiles )
     {

          int group_index = in_groupindexvector[tid];
          int index_in_trainedMatrix = group_index*4;
          int im=0, iv=1;
          float bprob=0.0, mprob=0.0;

          __syncthreads();

          for( int i=0; i<in_numopcode; i++)
          {
               float x;
               x = in_testMatrix[ i*in_numtestfiles + tid ];
               if( x > 0) 
               {
                    float bvar, bmean, mvar, mmean;
                    bmean = in_trainedMatrix[ (0+im+index_in_trainedMatrix)*in_numopcode + i];
                    bvar  = in_trainedMatrix[ (0+iv+index_in_trainedMatrix)*in_numopcode + i];

                    bprob += getTheProbablityD( x, bmean, bvar);

                    mmean = in_trainedMatrix[ (2+im+index_in_trainedMatrix)*in_numopcode+ i];
                    mvar  = in_trainedMatrix[ (2+iv+index_in_trainedMatrix)*in_numopcode+ i];
                    mprob += getTheProbablityD( x, mmean, mvar);
               }

          }
          __syncthreads();

          if( bprob > mprob) 
               out_predictvector[ tid ] = 0;
          else
               out_predictvector[ tid ] = 1;
     }
     __syncthreads();
}

void passignClassUsingMeanVarianceData( 
          float *in_trainedMatrix,
          float *in_testMatrix,
          int in_numgropus,
          int in_numopcode,
          int in_numtestfiles,
          int *in_groupindexvector,
          int *out_predictvector
          )
{
     /*
TODO: try checking if we can get speed up by making trainedMatrix go to 
constant memory
      */
     dim3 gridProp( ceil(in_numtestfiles/BLOCK_WIDTH)+1,1,1);
     dim3 blockProp(BLOCK_WIDTH,1,1);
     printf(" Running Kernel %d Threads.\n",BLOCK_WIDTH);
     printf(" Running %.0lf Blocks.\n",ceil(in_numtestfiles/BLOCK_WIDTH));
     err = hipSuccess;
     assignClassUsingMeanVarianceDataKernel<<<gridProp,blockProp>>>(
               in_trainedMatrix,
               in_testMatrix,
               in_numgropus,
               in_numopcode,
               in_numtestfiles,
               in_groupindexvector,
               out_predictvector
               );
     if (err != hipSuccess)
     {
          fprintf(stderr, "%s, %d.\n %s.", __FILE__, __LINE__, hipGetErrorString(err));
          exit(EXIT_FAILURE);
     }

}

     __global__
void assignClassUsingMeanVarianceDataUsingFeatureSelectionKernel(
          float *in_trainedMatrix,
          float *in_testMatrix,
          int *in_featureMatrix,
          int in_numgropus,
          int in_numopcode,
          int in_numtestfiles,
          int *in_groupindexvector,
          int *out_predictvector
          )
{
     int tid = threadIdx.x + blockIdx.x * BLOCK_WIDTH;
     int group_index = in_groupindexvector[tid];
     int index_in_trainedMatrix = group_index*4;
     int im=0, iv=1;
     float bprob=0.0, mprob=0.0;

     __syncthreads();

     for( int i=0; i<in_numopcode; i++)
     {
               int feature;
               float x;
               x = in_testMatrix[ i*in_numtestfiles + tid ];
               feature = in_featureMatrix[ group_index*in_numopcode+ i];
               if( x > 0) 
               {
                    float bvar, bmean, mvar, mmean;
                    bmean = in_trainedMatrix[ (0+im+index_in_trainedMatrix)*in_numopcode + i];
                    bvar  = in_trainedMatrix[ (0+iv+index_in_trainedMatrix)*in_numopcode + i];

                    bprob += getTheProbablityD( x, bmean, bvar)*feature;

                    mmean = in_trainedMatrix[ (2+im+index_in_trainedMatrix)*in_numopcode+ i];
                    mvar  = in_trainedMatrix[ (2+iv+index_in_trainedMatrix)*in_numopcode+ i];
                    mprob += getTheProbablityD( x, mmean, mvar)*feature;
               }
     }
     __syncthreads();

     if( bprob > mprob ) 
          out_predictvector[ tid ] = 0;
     else
          out_predictvector[ tid ] = 1;
}
void passignClassUsingMeanVarianceDataUsingFeatureSelection( 
          float *in_trainedMatrix, 
          float *in_testMatrix, 
          int *in_featureMatrix,
          int in_numgroups, 
          int in_numopcode, 
          int in_numtestfiles, 
          int *in_groupindexvector, 
          int *out_predictvector
          )
{
     /*
TODO: try checking if we can get speed up by making trainedMatrix go to 
constant memory
      */
     dim3 gridProp( ceil(in_numtestfiles/BLOCK_WIDTH)+1,1,1);
     dim3 blockProp(BLOCK_WIDTH,1,1);
     printf(" Running Kernel %d Threads.\n",BLOCK_WIDTH);
     printf(" Running %.0lf Blocks.\n",ceil(in_numtestfiles/BLOCK_WIDTH));
     err = hipSuccess;
     assignClassUsingMeanVarianceDataUsingFeatureSelectionKernel<<<gridProp,blockProp>>>(
               in_trainedMatrix,
               in_testMatrix,
               in_featureMatrix,
               in_numgroups,
               in_numopcode,
               in_numtestfiles,
               in_groupindexvector,
               out_predictvector
               );
     if (err != hipSuccess)
     {
          fprintf(stderr, "%s, %d.\n %s.", __FILE__, __LINE__, hipGetErrorString(err));
          exit(EXIT_FAILURE);
     }

}
