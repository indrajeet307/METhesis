#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "gpu_naive.h"
#define BLOCK_WIDTH 64
hipError_t err;
void createDeviceMatrixF(float **mat, int rows, int columns)
{
    err = hipSuccess;
    err = hipMalloc( mat, rows*columns*sizeof(float) );
    if ( err != hipSuccess )
    {
        fprintf(stderr, "#Error %s, %d.\n%s.", __FILE__, __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMemset((*mat), 0, rows*columns*sizeof(float));
    if ( err != hipSuccess )
    {
        fprintf(stderr, "#Error %s, %d.\n%s.", __FILE__, __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

void createDeviceMatrixI(int **mat, int rows, int columns)
{
    err = hipSuccess;
    err = hipMalloc( mat, rows*columns*sizeof(int) );
    if ( err != hipSuccess )
    {
        fprintf(stderr, "#Error %s, %d.\n%s.", __FILE__, __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMemset((*mat), 0, rows*columns*sizeof(int));
    if ( err != hipSuccess )
    {
        fprintf(stderr, "#Error %s, %d.\n%s.", __FILE__, __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

void transferToDeviceI(int *hostptr, int *deviceptr, int size)
{
    err = hipSuccess;
    err = hipMemcpy(deviceptr, hostptr, sizeof(float)*size, hipMemcpyHostToDevice);
    if(err != hipSuccess)
    {
        fprintf(stderr, "#Error %s, %d.\n%s.", __FILE__, __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

void transferToDeviceF(float *hostptr, float *deviceptr, int size)
{
    err = hipSuccess;
    err = hipMemcpy(deviceptr, hostptr, sizeof(float)*size, hipMemcpyHostToDevice);
    if(err != hipSuccess)
    {
        fprintf(stderr, "#Error %s, %d.\n%s.", __FILE__, __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

void transferFromDeviceI(int *hostptr, int *deviceptr, int size)
{
    err = hipSuccess;
    err = hipMemcpy(hostptr, deviceptr, sizeof(int)*size, hipMemcpyDeviceToHost);
    if(err != hipSuccess)
    {
        fprintf(stderr, "#Error %s, %d.\n%s.", __FILE__, __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

void transferFromDeviceF(float *hostptr, float *deviceptr, int size)
{
    err = hipSuccess;
    err = hipMemcpy(hostptr, deviceptr, sizeof(float)*size, hipMemcpyDeviceToHost);
    if(err != hipSuccess)
    {
        fprintf(stderr, "#Error %s, %d.\n%s.", __FILE__, __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

/*
   files along x and opcodes along y
 */
    __global__
void naiveTrainKernel(int *inmat, int *inclass, int *class_wise,float *outmat, float *class_prob, int inrows, int incolumns, int outrows)
{
    int index = blockIdx.x*BLOCK_WIDTH+threadIdx.x;
    int cls;
    if( index < incolumns)
    {
        cls = inclass[ index ];

        for ( int i=0; i<inrows; i++)
        {
            atomicAdd( &(class_wise[ cls ]), inmat[ i*incolumns + index]);
            atomicAdd( &(outmat[ cls*inrows + i]), (float)inmat[ i*incolumns + index]);
            atomicAdd( &(class_prob[ cls ]), (float)1);
        }
        __syncthreads();
        for ( int i=0; i<inrows; i++)
        {
            float temp = ( log10(( outmat[ cls*inrows+ i]+1)/ (class_wise[ cls ]+1)));
            outmat[ cls*inrows+ i] = (-1)*temp;
        }
        __syncthreads();
        class_prob[ cls ] = (-1)*log10( class_prob[ cls ]/incolumns );
    }
}

void pnaiveTrain( int *inmat, int *inclass, int *class_wise, float *outmat, float *class_prob, int inrows, int incolumns, int outrows)
{
    dim3 gridProp( ceil(incolumns/BLOCK_WIDTH)+1,1,1);
    dim3 blockProp(BLOCK_WIDTH,1,1);
    printf(" Running %d Threads.\n",BLOCK_WIDTH);
    printf(" Running %.0lf Blocks.\n",ceil(incolumns/BLOCK_WIDTH));
    err = hipSuccess;
    naiveTrainKernel<<<gridProp,blockProp>>>( inmat, inclass, class_wise, outmat, class_prob, inrows, incolumns, outrows);
    if (err != hipSuccess)
    {
        fprintf(stderr, "%s, %d.\n %s.", __FILE__, __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

__global__

void naiveTestKernel( float *in_probmat, float *in_class_prob, int*in_test_mat,
                 int in_test_mat_columns, int in_test_mat_rows, int in_probmat_rows, 
                 int in_probmat_columns, int *out_assigned_class)
{
    int index = blockIdx.x*BLOCK_WIDTH+threadIdx.x;
    float cls[2];
    if( index < in_test_mat_columns)
    {
        cls[0] = cls[1] =0;
        for( int i =0 ;i<in_test_mat_rows;i++)
        {
            if( in_test_mat[ i*in_test_mat_columns+index ] > 0)
            {
                for ( int j=0; j<in_probmat_rows; j++)
                {
                    cls[j] += (float)in_test_mat[ i*in_test_mat_columns+index]*in_probmat[
                    j*in_probmat_columns+ i];
                }
            }
        }
        out_assigned_class[ index ] = cls[0] > cls[1] ? 0 : 1;
    }
}

void pnaiveTest( float *in_probmat, float *in_class_prob, int*in_test_mat,
                 int in_test_mat_columns, int in_test_mat_rows, int in_probmat_rows, 
                 int in_probmat_columns, int *out_assigned_class)
{
    dim3 gridProp( ceil(in_test_mat_columns/BLOCK_WIDTH)+1,1,1);
    dim3 blockProp(BLOCK_WIDTH,1,1);
    printf(" Running %d Threads.\n",BLOCK_WIDTH);
    printf(" Running %.0lf Blocks.\n",ceil(in_test_mat_columns/BLOCK_WIDTH));
    err = hipSuccess;
    naiveTestKernel<<<gridProp,blockProp>>>(in_probmat, in_class_prob, in_test_mat,
    in_test_mat_columns, in_test_mat_rows, in_probmat_rows, 
    in_probmat_columns, out_assigned_class);
    if (err != hipSuccess)
    {
        fprintf(stderr, "%s, %d.\n %s.", __FILE__, __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}
